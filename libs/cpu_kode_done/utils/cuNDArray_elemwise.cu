#include "cuNDArray_elemwise.h"
#include "cuNDArray_operators.h"
#include "complext.h"

#include <complex>
#include <thrust/functional.h>

using namespace std;
 
namespace Gadgetron{

  template<typename T> struct cuNDA_real : public thrust::unary_function<T,typename realType<T>::Type>
  {
    __device__ typename realType<T>::Type operator()(const T &x) const {return real(x);}
  };
  
  template<class T> boost::shared_ptr< cuNDArray<typename realType<T>::Type> > 
  real( cuNDArray<T> *x )
  { 
    if( x == 0x0 )
      throw std::runtime_error("Gadgetron::real(): Invalid input array");
    
    boost::shared_ptr< cuNDArray<typename realType<T>::Type> > result(new cuNDArray<typename realType<T>::Type>());
    result->create(x->get_dimensions());
    thrust::device_ptr<typename realType<T>::Type> resPtr = result->get_device_ptr();
    thrust::device_ptr<T> xPtr = x->get_device_ptr();
    thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_real<T>());
    return result;
  }
  
  template <typename T> struct cuNDA_imag : public thrust::unary_function<T,typename realType<T>::Type>
  {
    __device__ typename realType<T>::Type operator()(const T &x) const {return imag(x);}
  };
  
  template<class T> boost::shared_ptr< cuNDArray<typename realType<T>::Type> > 
  imag( cuNDArray<T> *x )
  { 
    if( x == 0x0 )
      throw std::runtime_error("Gadgetron::imag(): Invalid input array");
    
    boost::shared_ptr< cuNDArray<typename realType<T>::Type> > result(new cuNDArray<typename realType<T>::Type>());
    result->create(x->get_dimensions());
    thrust::device_ptr<typename realType<T>::Type> resPtr = result->get_device_ptr();
    thrust::device_ptr<T> xPtr = x->get_device_ptr();
    thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_imag<T>());
    return result;
  }
  
  
  template <typename T> struct cuNDA_real_to_complex : public thrust::unary_function<typename realType<T>::Type,T>
  {
    __device__ T operator()(const typename realType<T>::Type &x) const {return T(x);}
  };
  
  template<class T> boost::shared_ptr< cuNDArray<T> > 
  real_to_complex( cuNDArray<typename realType<T>::Type> *x )
  {
    if( x == 0x0 )
      throw std::runtime_error("Gadgetron::real_to_complex(): Invalid input array");
    
    boost::shared_ptr< cuNDArray<T> > result(new cuNDArray<T>());
    result->create(x->get_dimensions());
    thrust::device_ptr<T> resPtr = result->get_device_ptr();
    thrust::device_ptr<typename realType<T>::Type> xPtr = x->get_device_ptr();
    thrust::transform(xPtr,xPtr+x->get_number_of_elements(),resPtr,cuNDA_real_to_complex<T>());
    return result;
  }

  //
  // Instantiation
  //
  
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > real<float_complext>( cuNDArray<float_complext>* );
  
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > imag<float_complext>( cuNDArray<float_complext>* );
  
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > real_to_complex<float_complext>( cuNDArray<float>* );
}

