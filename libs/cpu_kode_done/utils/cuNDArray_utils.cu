#include "hip/hip_runtime.h"
#include "cuNDArray_utils.h"
#include "vector_td_utilities.h"
#include "cudaDeviceManager.h"
#include "setup_grid.h"

#include <cmath>

namespace Gadgetron {

  template <class T> 
  __global__ void cuNDArray_permute_kernel(
					   T* in, T* out, 
					   unsigned int ndim,
					   unsigned int* dims,
					   unsigned int* strides_out,
					   unsigned long int elements,
					   int shift_mode)
  {
    unsigned long idx_in = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
    unsigned long idx_out = 0;
    unsigned long idx_in_tmp = idx_in;

    if (idx_in < elements) {

      unsigned int cur_index;
      for (unsigned int i = 0; i < ndim; i++) {
	unsigned long idx_in_remainder = idx_in_tmp / dims[i];
	cur_index = idx_in_tmp-(idx_in_remainder*dims[i]); //cur_index = idx_in_tmp%dims[i];
	if (shift_mode < 0) { //IFFTSHIFT
	  idx_out += ((cur_index+(dims[i]>>1))%dims[i])*strides_out[i];
	} else if (shift_mode > 0) { //FFTSHIFT
	  idx_out += ((cur_index+((dims[i]+1)>>1))%dims[i])*strides_out[i];
	} else {
	  idx_out += cur_index*strides_out[i];
	}
	idx_in_tmp = idx_in_remainder;
      }
      out[idx_in] = in[idx_out];
    }
  }

  template <class T> void cuNDArray_permute(cuNDArray<T>* in,
					    cuNDArray<T>* out,
					    std::vector<unsigned int> *order,
					    int shift_mode)
  {
      
    if( out == 0x0 ){
      throw cuda_error("cuNDArray_permute(internal): 0x0 output");;
    }

    hipError_t err;

    T* in_ptr = in->get_data_ptr();
    T* out_ptr = 0;

    if (out) {
      out_ptr = out->get_data_ptr();
    } else {
      if (hipMalloc((void**) &out_ptr, in->get_number_of_elements()*sizeof(T)) != hipSuccess) {
	throw cuda_error("cuNDArray_permute : Error allocating CUDA memory");;
      }
    }

    unsigned int* dims        = new unsigned int[in->get_number_of_dimensions()];
    unsigned int* strides_out = new unsigned int[in->get_number_of_dimensions()];

    if (!dims || !strides_out) {
      throw cuda_error("cuNDArray_permute: failed to allocate temporary storage for arrays");;
    }

    for (unsigned int i = 0; i < in->get_number_of_dimensions(); i++) {
      dims[i] = (*in->get_dimensions())[(*order)[i]];
      strides_out[i] = 1;    
      for (unsigned int j = 0; j < (*order)[i]; j++) {
	strides_out[i] *= (*in->get_dimensions())[j];
      }
    }

    unsigned int* dims_dev        = 0;
    unsigned int* strides_out_dev = 0;

    if (hipMalloc((void**) &dims_dev, in->get_number_of_dimensions()*sizeof(unsigned int)) != hipSuccess) {
      throw cuda_error("cuNDArray_permute : Error allocating CUDA dims memory");;
    }

    if (hipMalloc((void**) &strides_out_dev, in->get_number_of_dimensions()*sizeof(unsigned int)) != hipSuccess) {
      throw cuda_error("cuNDArray_permute : Error allocating CUDA strides_out memory");;
    }

    if (hipMemcpy(dims_dev, dims, in->get_number_of_dimensions()*sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) {
      err = hipGetLastError();
      std::stringstream ss;
      ss << "cuNDArray_permute : Error uploading dimensions to device, " << hipGetErrorString(err);
      throw cuda_error(ss.str());;
    }

    if (hipMemcpy(strides_out_dev, strides_out, in->get_number_of_dimensions()*sizeof(unsigned int), hipMemcpyHostToDevice) != hipSuccess) {
      throw cuda_error("cuNDArray_permute : Error uploading strides to device");;
    }

    dim3 blockDim(512,1,1);
    dim3 gridDim;
    if( in->get_number_of_dimensions() > 2 ){
      gridDim = dim3((unsigned int) std::ceil((double)in->get_size(0)*in->get_size(1)/blockDim.x), 1, 1 );
      for( unsigned int d=2; d<in->get_number_of_dimensions(); d++ )
	gridDim.y *= in->get_size(d);
    }
    else
      gridDim = dim3((unsigned int) std::ceil((double)in->get_number_of_elements()/blockDim.x), 1, 1 );

    cuNDArray_permute_kernel<<< gridDim, blockDim >>>( in_ptr, out_ptr, in->get_number_of_dimensions(), 
						       dims_dev, strides_out_dev, in->get_number_of_elements(), shift_mode);

    err = hipGetLastError();
    if( err != hipSuccess ){
      std::stringstream ss;
      ss <<"cuNDArray_permute : Error during kernel call: " << hipGetErrorString(err);
      throw cuda_error(ss.str());;
    }

    if (hipFree(dims_dev) != hipSuccess) {
      err = hipGetLastError();
      std::stringstream ss;
      ss << "cuNDArray_permute: failed to delete device memory (dims_dev) " << hipGetErrorString(err);
      throw cuda_error(ss.str());;
    }

    if (hipFree(strides_out_dev) != hipSuccess) {
      err = hipGetLastError();
      std::stringstream ss;
      ss << "cuNDArray_permute: failed to delete device memory (strides_out_dev) "<< hipGetErrorString(err);
      throw cuda_error(ss.str());;
    }    
    delete [] dims;
    delete [] strides_out;    
  }  

  template <class T> boost::shared_ptr< cuNDArray<T> >
  permute( cuNDArray<T> *in, std::vector<unsigned int> *dim_order, int shift_mode )
  {
    if( in == 0x0 || dim_order == 0x0 ) {
      throw std::runtime_error("permute(): invalid pointer provided");;
    }    

    std::vector<unsigned int> dims;
    for (unsigned int i = 0; i < dim_order->size(); i++)
      dims.push_back(in->get_dimensions()->at(dim_order->at(i)));
    boost::shared_ptr< cuNDArray<T> > out( new cuNDArray<T>() );    
    out->create(&dims);
    permute( in, out.get(), dim_order, shift_mode );
    return out;
  }

  template <class T> void
  permute( cuNDArray<T> *in, cuNDArray<T> *out, std::vector<unsigned int> *dim_order, int shift_mode )
  {
    if( in == 0x0 || out == 0x0 || dim_order == 0x0 ) {
      throw std::runtime_error("permute(): invalid pointer provided");;
    }    

    //Check ordering array
    if (dim_order->size() > in->get_number_of_dimensions()) {
      throw std::runtime_error("permute(): invalid length of dimension ordering array");;
    }

    std::vector<unsigned int> dim_count(in->get_number_of_dimensions(),0);
    for (unsigned int i = 0; i < dim_order->size(); i++) {
      if ((*dim_order)[i] >= in->get_number_of_dimensions()) {
	throw std::runtime_error("permute(): invalid dimension order array");;
      }
      dim_count[(*dim_order)[i]]++;
    }

    //Create an internal array to store the dimensions
    std::vector<unsigned int> dim_order_int;

    //Check that there are no duplicate dimensions
    for (unsigned int i = 0; i < dim_order->size(); i++) {
      if (dim_count[(*dim_order)[i]] != 1) {
	throw std::runtime_error("permute(): invalid dimension order array (duplicates)");;
      }
      dim_order_int.push_back((*dim_order)[i]);
    }

    for (unsigned int i = 0; i < dim_order_int.size(); i++) {
      if ((*in->get_dimensions())[dim_order_int[i]] != out->get_size(i)) {
	throw std::runtime_error("permute(): dimensions of output array do not match the input array");;
      }
    }

    //Pad dimension order array with dimension not mentioned in order array
    if (dim_order_int.size() < in->get_number_of_dimensions()) {
      for (unsigned int i = 0; i < dim_count.size(); i++) {
	if (dim_count[i] == 0) {
	  dim_order_int.push_back(i);
	}
      }
    }    
    cuNDArray_permute(in, out, &dim_order_int, shift_mode);
  }

  template<class T> boost::shared_ptr< cuNDArray<T> >
  shift_dim( cuNDArray<T> *in, int shift )
  {
    if( in == 0x0 ) {
      throw std::runtime_error("shift_dim(): invalid input pointer provided");;
    }    

    std::vector<unsigned int> order;
    for (int i = 0; i < in->get_number_of_dimensions(); i++) {
      order.push_back(static_cast<unsigned int>((i+shift)%in->get_number_of_dimensions()));
    }
    return permute(in,&order);
  }

  template<class T> 
  void shift_dim( cuNDArray<T> *in, cuNDArray<T> *out, int shift )
  {
    if( in == 0x0 || out == 0x0 ) {
      throw std::runtime_error("shift_dim(): invalid pointer provided");;
    }    

    std::vector<unsigned int> order;
    for (int i = 0; i < in->get_number_of_dimensions(); i++) {
      order.push_back(static_cast<unsigned int>((i+shift)%in->get_number_of_dimensions()));
    }
    permute(in,out,&order);
  }

  template<class T> static void find_stride( cuNDArray<T> *in, unsigned int dim, unsigned int *stride, std::vector<unsigned int> *dims )
  {
    *stride = 1;
    for( unsigned int i=0; i<in->get_number_of_dimensions(); i++ ){
      if( i != dim )
	dims->push_back(in->get_size(i));
      if( i < dim )
	*stride *= in->get_size(i);
    }
  }

  // Expand
  //
  template<class T> 
  __global__ void expand_kernel( 
				T *in, T *out, 
				unsigned int number_of_elements_in, unsigned int number_of_elements_out, unsigned int new_dim_size )
  {
    const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;    
    if( idx < number_of_elements_out ){
      out[idx] = in[idx%number_of_elements_in];
    }
  }

  // Expand
  //
  template<class T> boost::shared_ptr< cuNDArray<T> > 
  expand( cuNDArray<T> *in, unsigned int new_dim_size )
  {
    unsigned int number_of_elements_out = in->get_number_of_elements()*new_dim_size;

    // Setup block/grid dimensions
    dim3 blockDim; dim3 gridDim;
    setup_grid( number_of_elements_out, &blockDim, &gridDim );

    // Find element stride
    std::vector<unsigned int> dims = *in->get_dimensions();
    dims.push_back(new_dim_size);

    // Invoke kernel
    boost::shared_ptr< cuNDArray<T> > out( new cuNDArray<T>());
    out->create(&dims);

    expand_kernel<T><<< gridDim, blockDim >>>( in->get_data_ptr(), out->get_data_ptr(), 
					       in->get_number_of_elements(), number_of_elements_out, new_dim_size );

    CHECK_FOR_CUDA_ERROR();    
    return out;
  }

  // Sum
  //
  template<class T> 
  __global__ void sum_kernel( 
			     T *in, T *out, 
			     unsigned int stride, unsigned int number_of_batches, unsigned int number_of_elements )
  {
    const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

    if( idx < number_of_elements ){

      unsigned int in_idx = (idx/stride)*stride*number_of_batches+(idx%stride);

      T val = in[in_idx];

      for( unsigned int i=1; i<number_of_batches; i++ ) 
	val += in[i*stride+in_idx];

      out[idx] = val; 
    }
  }

  // Sum
  //
  template<class T>  boost::shared_ptr< cuNDArray<T> > sum( cuNDArray<T> *in, unsigned int dim )
  {
    // Some validity checks
    if( !(in->get_number_of_dimensions()>1) ){
      throw std::runtime_error("sum: underdimensioned.");;
    }

    if( dim > in->get_number_of_dimensions()-1 ){
      throw std::runtime_error( "sum: dimension out of range.");;
    }

    unsigned int number_of_batches = in->get_size(dim);
    unsigned int number_of_elements = in->get_number_of_elements()/number_of_batches;

    // Setup block/grid dimensions
    dim3 blockDim; dim3 gridDim;
    setup_grid( number_of_elements, &blockDim, &gridDim );

    // Find element stride
    unsigned int stride; std::vector<unsigned int> dims;
    find_stride<T>( in, dim, &stride, &dims );

    // Invoke kernel
    boost::shared_ptr< cuNDArray<T> > out(new cuNDArray<T>());
    out->create(&dims);

    sum_kernel<T><<< gridDim, blockDim >>>( in->get_data_ptr(), out->get_data_ptr(), stride, number_of_batches, number_of_elements );

    CHECK_FOR_CUDA_ERROR();
    return out;
  }

  // Crop
  template<class T, unsigned int D> __global__ void crop_kernel( 
								vector_td<unsigned int,D> offset, vector_td<unsigned int,D> matrix_size_in, vector_td<unsigned int,D> matrix_size_out,
								T *in, T *out, unsigned int num_batches, unsigned int num_elements )
  {
    typedef vector_td<unsigned int,D> uintd;
    const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
    const unsigned int frame_offset = idx/num_elements;

    if( idx < num_elements*num_batches ){
      const uintd co = idx_to_co<D>( idx-frame_offset*num_elements, matrix_size_out );
      const uintd co_os = offset + co;
      const unsigned int in_idx = co_to_idx<D>(co_os, matrix_size_in)+frame_offset*prod(matrix_size_in);
      out[idx] = in[in_idx];
    }
  }

  // Crop
  template<class T, unsigned int D>
  void crop( typename uintd<D>::Type offset, cuNDArray<T> *in, cuNDArray<T> *out )
  {
    if( in == 0x0 || out == 0x0 ){
      throw std::runtime_error("crop: 0x0 ndarray provided");;
    }

    if( in->get_number_of_dimensions() != out->get_number_of_dimensions() ){
      throw std::runtime_error("crop: image dimensions mismatch");;
    }

    if( in->get_number_of_dimensions() < D ){
      std::stringstream ss;
      ss << "crop: number of image dimensions should be at least " << D;
      throw std::runtime_error(ss.str());;
    }

    typename uintd<D>::Type matrix_size_in = from_std_vector<unsigned int,D>( *in->get_dimensions() );
    typename uintd<D>::Type matrix_size_out = from_std_vector<unsigned int,D>( *out->get_dimensions() );

    unsigned int number_of_batches = 1;
    for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
      number_of_batches *= in->get_size(d);
    }

    if( weak_greater(offset+matrix_size_out, matrix_size_in) ){
      throw std::runtime_error( "crop: cropping size mismatch");;
    }

    // Setup block/grid dimensions
    dim3 blockDim; dim3 gridDim;
    setup_grid( prod(matrix_size_out), &blockDim, &gridDim, number_of_batches );

    // Invoke kernel
    crop_kernel<T,D><<< gridDim, blockDim >>>
      ( offset, matrix_size_in, matrix_size_out, in->get_data_ptr(), out->get_data_ptr(), number_of_batches, prod(matrix_size_out) );

    CHECK_FOR_CUDA_ERROR();
  }

  template<class T, unsigned int D> boost::shared_ptr< cuNDArray<T> > 
  crop( typename uintd<D>::Type offset, typename uintd<D>::Type size, cuNDArray<T> *in )
  {
    if( in == 0x0 ){
      throw std::runtime_error("crop: 0x0 array provided");;
    }
    std::vector<unsigned int> dims = to_std_vector(size);
    for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
      dims.push_back(in->get_size(d));
    }
    boost::shared_ptr< cuNDArray<T> > result( new cuNDArray<T>(&dims) );
    crop<T,D>(offset, in, result.get());
    return result;
  }  

  // Expand and zero fill
  template<class T, unsigned int D> 
  __global__ void pad_kernel( 
			     vector_td<unsigned int,D> matrix_size_in, vector_td<unsigned int,D> matrix_size_out,
			     T *in, T *out, unsigned int number_of_batches, unsigned int num_elements, T val )
  {
    typedef vector_td<unsigned int,D> uintd;
    const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
    const unsigned int frame_offset = idx/num_elements;

    if( idx < num_elements*number_of_batches ){

      const uintd co_out = idx_to_co<D>( idx-frame_offset*num_elements, matrix_size_out );
      const uintd offset = (matrix_size_out-matrix_size_in)>>1;
      T _out;
      bool inside = (co_out>=offset) && (co_out<(matrix_size_in+offset));

      if( inside )
	_out = in[co_to_idx<D>(co_out-offset, matrix_size_in)+frame_offset*prod(matrix_size_in)];
      else{      
	_out = val;
      }

      out[idx] = _out;
    }
  }

  template<class T, unsigned int D> 
  void pad( cuNDArray<T> *in, cuNDArray<T> *out, T val )
  { 
    if( in == 0x0 || out == 0x0 ){
      throw std::runtime_error("pad: 0x0 ndarray provided");;
    }

    if( in->get_number_of_dimensions() != out->get_number_of_dimensions() ){
      throw std::runtime_error("pad: image dimensions mismatch");;
    }

    if( in->get_number_of_dimensions() < D ){
      std::stringstream ss;
      ss << "pad: number of image dimensions should be at least " << D;
      throw std::runtime_error(ss.str());;
    }

    typename uintd<D>::Type matrix_size_in = from_std_vector<unsigned int,D>( *in->get_dimensions() );
    typename uintd<D>::Type matrix_size_out = from_std_vector<unsigned int,D>( *out->get_dimensions() );

    unsigned int number_of_batches = 1;
    for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
      number_of_batches *= in->get_size(d);
    }

    if( weak_greater(matrix_size_in,matrix_size_out) ){
      throw std::runtime_error("pad: size mismatch, cannot expand");
    }

    // Setup block/grid dimensions
    dim3 blockDim; dim3 gridDim;
    setup_grid( prod(matrix_size_out), &blockDim, &gridDim, number_of_batches );

    // Invoke kernel
    pad_kernel<T,D><<< gridDim, blockDim >>> 
      ( matrix_size_in, matrix_size_out, in->get_data_ptr(), out->get_data_ptr(), number_of_batches, prod(matrix_size_out), val );

    CHECK_FOR_CUDA_ERROR();
  }

  template<class T, unsigned int D> boost::shared_ptr< cuNDArray<T> >
  pad( typename uintd<D>::Type size, cuNDArray<T> *in, T val )
  {
    if( in == 0x0 ){
      throw std::runtime_error("pad: 0x0 array provided");;
    }
    std::vector<unsigned int> dims = to_std_vector(size);
    for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
      dims.push_back(in->get_size(d));
    }
    boost::shared_ptr< cuNDArray<T> > result( new cuNDArray<T>(&dims) );
    pad<T,D>(in, result.get(), val);
    return result;
  }

  template<class T, unsigned int D> 
  __global__ void fill_border_kernel( 
				     vector_td<unsigned int,D> matrix_size_in, vector_td<unsigned int,D> matrix_size_out,
				     T *image, unsigned int number_of_batches, unsigned int number_of_elements, T val )
  {
    const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

    if( idx < number_of_elements ){
      const vector_td<unsigned int,D> co_out = idx_to_co<D>( idx, matrix_size_out );
      const vector_td<unsigned int,D> offset = (matrix_size_out-matrix_size_in)>>1;
      if( weak_less( co_out, offset ) || weak_greater_equal( co_out, matrix_size_in+offset ) ){
	for( unsigned int batch=0; batch<number_of_batches; batch++ ){
	  image[idx+batch*number_of_elements] = val;
	}
      }
      else
	; // do nothing
    }
  }

  // Zero fill border (rectangular)
  template<class T, unsigned int D> 
  void fill_border( typename uintd<D>::Type matrix_size_in, cuNDArray<T> *in_out, T val )
  { 
    typename uintd<D>::Type matrix_size_out = from_std_vector<unsigned int,D>( *in_out->get_dimensions() );

    if( weak_greater(matrix_size_in, matrix_size_out) ){
      throw std::runtime_error("fill_border: size mismatch, cannot zero fill");;
    }

    unsigned int number_of_batches = 1;
    for( unsigned int d=D; d<in_out->get_number_of_dimensions(); d++ ){
      number_of_batches *= in_out->get_size(d);
    }

    // Setup block/grid dimensions
    dim3 blockDim; dim3 gridDim;
    setup_grid( prod(matrix_size_out), &blockDim, &gridDim );

    // Invoke kernel
    fill_border_kernel<T,D><<< gridDim, blockDim >>>
      ( matrix_size_in, matrix_size_out, in_out->get_data_ptr(), number_of_batches, prod(matrix_size_out), val );

    CHECK_FOR_CUDA_ERROR();
  }


  template<class T> T mean(cuNDArray<T>* in)
  {
    return thrust::reduce(in->begin(),in->end(),T(0),thrust::plus<T>())/T(in->get_number_of_elements());
  }

  template<class T> T min(cuNDArray<T>* in)
	{
  	return *thrust::min_element(in->begin(),in->end());
	}

  template<class T> T max(cuNDArray<T>* in)
	{
		return *thrust::max_element(in->begin(),in->end());
	}

  //
  // Instantiation
  //

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > permute( cuNDArray<float>*, std::vector<unsigned int>*, int );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > permute( cuNDArray<double>*, std::vector<unsigned int>*, int );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > permute( cuNDArray<float_complext>*, std::vector<unsigned int>*, int );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > permute( cuNDArray<double_complext>*, std::vector<unsigned int>*, int );  

  template EXPORTGPUCORE void permute( cuNDArray<float>*, cuNDArray<float>*, std::vector<unsigned int>*, int);
  template EXPORTGPUCORE void permute( cuNDArray<double>*, cuNDArray<double>*, std::vector<unsigned int>*, int);
  template EXPORTGPUCORE void permute( cuNDArray<float_complext>*, cuNDArray<float_complext>*, std::vector<unsigned int>*, int);
  template EXPORTGPUCORE void permute( cuNDArray<double_complext>*, cuNDArray<double_complext>*, std::vector<unsigned int>*, int);

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > shift_dim( cuNDArray<float>*, int );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > shift_dim( cuNDArray<double>*, int );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > shift_dim( cuNDArray<float_complext>*, int );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > shift_dim( cuNDArray<double_complext>*, int );

  template EXPORTGPUCORE void shift_dim( cuNDArray<float>*, cuNDArray<float>*, int shift );
  template EXPORTGPUCORE void shift_dim( cuNDArray<double>*, cuNDArray<double>*, int shift );
  template EXPORTGPUCORE void shift_dim( cuNDArray<float_complext>*, cuNDArray<float_complext>*, int shift );
  template EXPORTGPUCORE void shift_dim( cuNDArray<double_complext>*, cuNDArray<double_complext>*, int shift );

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > expand<float>( cuNDArray<float>*, unsigned int);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > expand<double>( cuNDArray<double>*, unsigned int);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > expand<float_complext>( cuNDArray<float_complext>*, unsigned int);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > expand<double_complext>( cuNDArray<double_complext>*, unsigned int);

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > sum<float>( cuNDArray<float>*, unsigned int);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > sum<double>( cuNDArray<double>*, unsigned int);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > sum<float_complext>( cuNDArray<float_complext>*, unsigned int);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > sum<double_complext>( cuNDArray<double_complext>*, unsigned int);  

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > crop<float,1>( typename uintd<1>::Type, typename uintd<1>::Type, cuNDArray<float>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > crop<float,2>( typename uintd<2>::Type, typename uintd<2>::Type, cuNDArray<float>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > crop<float,3>( typename uintd<3>::Type, typename uintd<3>::Type, cuNDArray<float>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > crop<float,4>( typename uintd<4>::Type, typename uintd<4>::Type, cuNDArray<float>*);

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > crop<float_complext,1>( typename uintd<1>::Type, typename uintd<1>::Type, cuNDArray<float_complext>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > crop<float_complext,2>( typename uintd<2>::Type, typename uintd<2>::Type, cuNDArray<float_complext>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > crop<float_complext,3>( typename uintd<3>::Type, typename uintd<3>::Type, cuNDArray<float_complext>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > crop<float_complext,4>( typename uintd<4>::Type, typename uintd<4>::Type, cuNDArray<float_complext>*);

  template EXPORTGPUCORE void crop<float,1>( uintd1, cuNDArray<float>*, cuNDArray<float>*);
  template EXPORTGPUCORE void crop<float,2>( uintd2, cuNDArray<float>*, cuNDArray<float>*);
  template EXPORTGPUCORE void crop<float,3>( uintd3, cuNDArray<float>*, cuNDArray<float>*);
  template EXPORTGPUCORE void crop<float,4>( uintd4, cuNDArray<float>*, cuNDArray<float>*);

  template EXPORTGPUCORE void crop<complext<float>,1>( uintd1, cuNDArray<complext<float> >*, cuNDArray< complext<float> >*);
  template EXPORTGPUCORE void crop<complext<float>,2>( uintd2, cuNDArray<complext<float> >*, cuNDArray< complext<float> >*);
  template EXPORTGPUCORE void crop<complext<float>,3>( uintd3, cuNDArray<complext<float> >*, cuNDArray< complext<float> >*);
  template EXPORTGPUCORE void crop<complext<float>,4>( uintd4, cuNDArray<complext<float> >*, cuNDArray< complext<float> >*);

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > pad<float,1>( typename uintd<1>::Type, cuNDArray<float>*, float );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > pad<float,2>( typename uintd<2>::Type, cuNDArray<float>*, float );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > pad<float,3>( typename uintd<3>::Type, cuNDArray<float>*, float );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> > pad<float,4>( typename uintd<4>::Type, cuNDArray<float>*, float );

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > pad<float_complext,1>( typename uintd<1>::Type, cuNDArray<float_complext>*, float_complext );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > pad<float_complext,2>( typename uintd<2>::Type, cuNDArray<float_complext>*, float_complext );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > pad<float_complext,3>( typename uintd<3>::Type, cuNDArray<float_complext>*, float_complext );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> > pad<float_complext,4>( typename uintd<4>::Type, cuNDArray<float_complext>*, float_complext );

  template EXPORTGPUCORE void pad<float,1>( cuNDArray<float>*, cuNDArray<float>*, float);
  template EXPORTGPUCORE void pad<float,2>( cuNDArray<float>*, cuNDArray<float>*, float);
  template EXPORTGPUCORE void pad<float,3>( cuNDArray<float>*, cuNDArray<float>*, float);
  template EXPORTGPUCORE void pad<float,4>( cuNDArray<float>*, cuNDArray<float>*, float);

  template EXPORTGPUCORE float mean<float>(cuNDArray<float>*);
  template EXPORTGPUCORE float_complext mean<float_complext>(cuNDArray<float_complext>*);

  template EXPORTGPUCORE float min<float>(cuNDArray<float>*);
  template EXPORTGPUCORE float max<float>(cuNDArray<float>*);


  template EXPORTGPUCORE void pad<float_complext,1>( cuNDArray<float_complext>*, cuNDArray<float_complext>*, float_complext);
  template EXPORTGPUCORE void pad<float_complext,2>( cuNDArray<float_complext>*, cuNDArray<float_complext>*, float_complext);  
  template EXPORTGPUCORE void pad<float_complext,3>( cuNDArray<float_complext>*, cuNDArray<float_complext>*, float_complext);
  template EXPORTGPUCORE void pad<float_complext,4>( cuNDArray<float_complext>*, cuNDArray<float_complext>*, float_complext);

  template EXPORTGPUCORE void fill_border<float,1>(uintd1, cuNDArray<float>*,float);
  template EXPORTGPUCORE void fill_border<float,2>(uintd2, cuNDArray<float>*,float);
  template EXPORTGPUCORE void fill_border<float,3>(uintd3, cuNDArray<float>*,float);
  template EXPORTGPUCORE void fill_border<float,4>(uintd4, cuNDArray<float>*,float);

  template EXPORTGPUCORE void fill_border<float_complext,1>(uintd1, cuNDArray<float_complext>*,float_complext);
  template EXPORTGPUCORE void fill_border<float_complext,2>(uintd2, cuNDArray<float_complext>*,float_complext);
  template EXPORTGPUCORE void fill_border<float_complext,3>(uintd3, cuNDArray<float_complext>*,float_complext);
  template EXPORTGPUCORE void fill_border<float_complext,4>(uintd4, cuNDArray<float_complext>*,float_complext);

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > crop<double,1>( typename uintd<1>::Type, typename uintd<1>::Type, cuNDArray<double>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > crop<double,2>( typename uintd<2>::Type, typename uintd<2>::Type, cuNDArray<double>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > crop<double,3>( typename uintd<3>::Type, typename uintd<3>::Type, cuNDArray<double>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > crop<double,4>( typename uintd<4>::Type, typename uintd<4>::Type, cuNDArray<double>*);

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > crop<double_complext,1>( typename uintd<1>::Type, typename uintd<1>::Type, cuNDArray<double_complext>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > crop<double_complext,2>( typename uintd<2>::Type, typename uintd<2>::Type, cuNDArray<double_complext>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > crop<double_complext,3>( typename uintd<3>::Type, typename uintd<3>::Type, cuNDArray<double_complext>*);
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > crop<double_complext,4>( typename uintd<4>::Type, typename uintd<4>::Type, cuNDArray<double_complext>*);

  template EXPORTGPUCORE void crop<double,1>( uintd1, cuNDArray<double>*, cuNDArray<double>*);
  template EXPORTGPUCORE void crop<double,2>( uintd2, cuNDArray<double>*, cuNDArray<double>*);
  template EXPORTGPUCORE void crop<double,3>( uintd3, cuNDArray<double>*, cuNDArray<double>*);
  template EXPORTGPUCORE void crop<double,4>( uintd4, cuNDArray<double>*, cuNDArray<double>*);

  template EXPORTGPUCORE void crop<complext<double>,1>( uintd1, cuNDArray<complext<double> >*, cuNDArray< complext<double> >*);
  template EXPORTGPUCORE void crop<complext<double>,2>( uintd2, cuNDArray<complext<double> >*, cuNDArray< complext<double> >*);
  template EXPORTGPUCORE void crop<complext<double>,3>( uintd3, cuNDArray<complext<double> >*, cuNDArray< complext<double> >*);
  template EXPORTGPUCORE void crop<complext<double>,4>( uintd4, cuNDArray<complext<double> >*, cuNDArray< complext<double> >*);

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > pad<double,1>( typename uintd<1>::Type, cuNDArray<double>*, double );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > pad<double,2>( typename uintd<2>::Type, cuNDArray<double>*, double );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > pad<double,3>( typename uintd<3>::Type, cuNDArray<double>*, double );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> > pad<double,4>( typename uintd<4>::Type, cuNDArray<double>*, double );

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > pad<double_complext,1>( typename uintd<1>::Type, cuNDArray<double_complext>*, double_complext );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > pad<double_complext,2>( typename uintd<2>::Type, cuNDArray<double_complext>*, double_complext );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > pad<double_complext,3>( typename uintd<3>::Type, cuNDArray<double_complext>*, double_complext );
  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> > pad<double_complext,4>( typename uintd<4>::Type, cuNDArray<double_complext>*, double_complext );

  template EXPORTGPUCORE void pad<double,1>( cuNDArray<double>*, cuNDArray<double>*, double);
  template EXPORTGPUCORE void pad<double,2>( cuNDArray<double>*, cuNDArray<double>*, double);
  template EXPORTGPUCORE void pad<double,3>( cuNDArray<double>*, cuNDArray<double>*, double);
  template EXPORTGPUCORE void pad<double,4>( cuNDArray<double>*, cuNDArray<double>*, double);

  template EXPORTGPUCORE void pad<double_complext,1>( cuNDArray<double_complext>*, cuNDArray<double_complext>*, double_complext);
  template EXPORTGPUCORE void pad<double_complext,2>( cuNDArray<double_complext>*, cuNDArray<double_complext>*, double_complext);  
  template EXPORTGPUCORE void pad<double_complext,3>( cuNDArray<double_complext>*, cuNDArray<double_complext>*, double_complext);
  template EXPORTGPUCORE void pad<double_complext,4>( cuNDArray<double_complext>*, cuNDArray<double_complext>*, double_complext);

  template EXPORTGPUCORE void fill_border<double,1>(uintd1, cuNDArray<double>*,double);
  template EXPORTGPUCORE void fill_border<double,2>(uintd2, cuNDArray<double>*,double);
  template EXPORTGPUCORE void fill_border<double,3>(uintd3, cuNDArray<double>*,double);
  template EXPORTGPUCORE void fill_border<double,4>(uintd4, cuNDArray<double>*,double);

  template EXPORTGPUCORE void fill_border<double_complext,1>(uintd1, cuNDArray<double_complext>*,double_complext);
  template EXPORTGPUCORE void fill_border<double_complext,2>(uintd2, cuNDArray<double_complext>*,double_complext);
  template EXPORTGPUCORE void fill_border<double_complext,3>(uintd3, cuNDArray<double_complext>*,double_complext);
  template EXPORTGPUCORE void fill_border<double_complext,4>(uintd4, cuNDArray<double_complext>*,double_complext);

  template EXPORTGPUCORE double mean<double>(cuNDArray<double>*);
  template EXPORTGPUCORE double_complext mean<double_complext>(cuNDArray<double_complext>*);
  
  template EXPORTGPUCORE double min<double>(cuNDArray<double>*);
	template EXPORTGPUCORE double max<double>(cuNDArray<double>*);
  // We can probably instantiate these functionsfor many more types? E.g. arrays of floatd2. 
  // For now we just introduce what we have needed...
  //

  template EXPORTGPUCORE boost::shared_ptr< cuNDArray<floatd2> > expand<floatd2>( cuNDArray<floatd2>*, unsigned int);  
}
